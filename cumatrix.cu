#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <random>
#include <stdexcept>

#include "cumatrix.h"

std::ostream& operator << (std::ostream& out, const cumatrix& mat) {
  for (int i = 0; i < mat.rows(); i++) {
    for (int j = 0; j < mat.cols(); j++) {
      out << " " << mat(i, j);
    }
    out << std::endl;
  }
  return out;
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "Error: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) throw new std::runtime_error("");
  }
}

void cumatrix::fill_rand() {
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> dis((float)-1.0, 1.0);
  for (int i = 0; i < size(); i++)
    (*this)[i] = dis(gen);
}

float* cumatrix::get_device_pointer(bool copy) {
  if (!in_device) {
    gpuErrchk(hipMalloc((void **) &d_elemns, N * M * sizeof(value_type)));
    if (copy) gpuErrchk(hipMemcpy(d_elemns, elemns, N * M * sizeof(value_type), hipMemcpyHostToDevice));
    in_device = true;
  }
  return d_elemns;
}

void cumatrix::refresh_from_device() {
  if (in_device) gpuErrchk(hipMemcpy(elemns, d_elemns, N * M * sizeof(value_type), hipMemcpyDeviceToHost));
}

void cumatrix::refresh_to_device() {
  if (in_device) gpuErrchk(hipMemcpy(d_elemns, elemns, N * M * sizeof(value_type), hipMemcpyHostToDevice));
}

void cumatrix::release_device_data() {
  if (in_device) {
    gpuErrchk(hipFree(d_elemns));
    in_device = false;
  }
}

cumatrix operator*(cumatrix& a, cumatrix& b) {
  cumatrix output(a.rows(), b.cols());
  const float* d_a = a.get_device_pointer();
  const float* d_b = b.get_device_pointer();
  float* d_c = output.get_device_pointer(false);
  int N = a.rows(), M = b.cols(), K = a.cols();
  int lda=N, ldb=K, ldc=N;
  const float alpha = 1;
  const float beta = 0;

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha, d_a, lda, d_b, ldb, &beta, d_c, ldc);

  hipblasDestroy(handle);
  output.refresh_from_device();
  return output;
}